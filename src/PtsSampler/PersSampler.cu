#include "hip/hip_runtime.h"
//
// Created by ppwang on 2022/9/26.
//

#include "PersSampler.h"
#include "../Utils/Utils.h"
#define MAX_STACK_SIZE 48
#define MAX_OCT_INTERSECT_PER_RAY 1024
#define MAX_SAMPLE_PER_RAY 1024

#define OCC_WEIGHT_BASE 512
#define ABS_WEIGHT_THRES 0.01
#define REL_WEIGHT_THRES 0.1

#define OCC_ALPHA_BASE 32
#define ABS_ALPHA_THRES 0.02
#define REL_ALPHA_THRES 0.1

using Tensor = torch::Tensor;

__global__ void MarkVistNodeKernel(int n_rays,
                                   int* pts_idx_start_end,
                                   int* oct_indices,
                                   float* sampled_weights,
                                   float* sampled_alpha,
                                   int* visit_weight_adder,
                                   int* visit_alpha_adder,
                                   int* visit_mark,
                                   int* visit_cnt) {
  const int ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ray_idx >= n_rays) { return; }
  const int pts_idx_start = pts_idx_start_end[ray_idx * 2];
  const int pts_idx_end   = pts_idx_start_end[ray_idx * 2 + 1];
  if (pts_idx_start >= pts_idx_end) { return; }
  float max_weight = 0.f;
  float max_alpha = 0.f;
  for (int pts_idx = pts_idx_start; pts_idx < pts_idx_end; pts_idx++) {
    max_weight = fmaxf(max_weight, sampled_weights[pts_idx]);
    max_alpha = fmaxf(max_alpha, sampled_alpha[pts_idx]);
  }

  const float weight_thres = fminf(max_weight * REL_WEIGHT_THRES, ABS_WEIGHT_THRES);
  const float alpha_thres = fminf(max_alpha * REL_ALPHA_THRES, ABS_ALPHA_THRES);

  float cur_oct_weight = 0.f;
  float cur_oct_alpha = 0.f;
  int cur_oct_idx = -1;
  int cur_visit_cnt = 0;
  for (int pts_idx = pts_idx_start; pts_idx < pts_idx_end; pts_idx++) {
    if (cur_oct_idx != oct_indices[pts_idx]) {
      if (cur_oct_idx >= 0) {
        atomicMax(visit_weight_adder + cur_oct_idx, cur_oct_weight > weight_thres ? OCC_WEIGHT_BASE : -1);
        atomicMax(visit_alpha_adder + cur_oct_idx, cur_oct_alpha > alpha_thres ? OCC_ALPHA_BASE : -1);
        atomicMax(visit_cnt + cur_oct_idx, cur_visit_cnt);
        visit_mark[cur_oct_idx] = 1;
      }
      cur_oct_idx = oct_indices[pts_idx];
      cur_oct_weight = 0.f;
      cur_oct_alpha = 0.f;
      cur_visit_cnt = 0;
    }
    cur_oct_weight = fmaxf(cur_oct_weight, sampled_weights[pts_idx]);
    cur_oct_alpha = fmaxf(cur_oct_alpha, sampled_alpha[pts_idx]);
    cur_visit_cnt += 1;
  }
  if (cur_oct_idx >= 0) {
    atomicMax(visit_weight_adder + cur_oct_idx, cur_oct_weight > weight_thres ? OCC_WEIGHT_BASE : -1);
    atomicMax(visit_alpha_adder + cur_oct_idx, cur_oct_alpha > alpha_thres ? OCC_ALPHA_BASE : -1);
    atomicMax(visit_cnt + cur_oct_idx, cur_visit_cnt);
    visit_mark[cur_oct_idx] = 1;
  }
}

__global__ void MarkInvalidNodes(int n_nodes, int* node_weight_stats, int* node_alpha_stats, TreeNode* nodes) {
  int oct_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (oct_idx >= n_nodes) { return; }
  if (node_weight_stats[oct_idx] < 0 || node_alpha_stats[oct_idx] < 0) {
    nodes[oct_idx].trans_idx = -1;
  }
}

__device__ int CheckVisible(const Wec3f& center, float side_len,
                            const Watrix33f& intri, const Watrix34f& w2c, const Wec2f& bound) {
  Wec3f cam_pt = w2c * center.homogeneous();
  float radius = side_len * 0.707;
  if (-cam_pt.z() < bound(0) - radius ||
      -cam_pt.z() > bound(1) + radius) {
    return 0;
  }
  if (cam_pt.norm() < radius) {
    return 1;
  }

  float cx = intri(0, 2);
  float cy = intri(1, 2);
  float fx = intri(0, 0);
  float fy = intri(1, 1);
  float bias_x = radius / -cam_pt.z() * fx;
  float bias_y = radius / -cam_pt.z() * fy;
  float img_pt_x = cam_pt.x() / -cam_pt.z() * fx;
  float img_pt_y = cam_pt.y() / -cam_pt.z() * fy;
  if (img_pt_x + bias_x < -cx || img_pt_x > cx + bias_x ||
      img_pt_y + bias_y < -cy || img_pt_y > cy + bias_y) {
    return 0;
  }
  return 1;
}

__global__ void MarkInvisibleNodesKernel(int n_nodes, int n_cams,
                                         TreeNode* tree_nodes,
                                         Watrix33f* intris, Watrix34f* w2cs, Wec2f* bounds) {
  int node_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (node_idx >= n_nodes) { return; }
  int n_visible_cams = 0;
  for (int cam_idx = 0; cam_idx < n_cams; cam_idx++) {
    n_visible_cams += CheckVisible(tree_nodes[node_idx].center,
                                   tree_nodes[node_idx].side_len,
                                   intris[cam_idx],
                                   w2cs[cam_idx],
                                   bounds[cam_idx]);
  }
  if (n_visible_cams < 1) {
    tree_nodes[node_idx].trans_idx = -1;
  }
}

void PersOctree::MarkInvisibleNodes() {
  int n_nodes = tree_nodes_.size();
  int n_cams = intri_.size(0);

  CK_CONT(intri_);
  CK_CONT(w2c_);
  CK_CONT(bound_);

  dim3 block_dim = LIN_BLOCK_DIM(n_nodes);
  dim3 grid_dim = LIN_GRID_DIM(n_nodes);
  MarkInvisibleNodesKernel<<<grid_dim, block_dim>>>(
      n_nodes, n_cams,
      RE_INTER(TreeNode*, tree_nodes_gpu_.data_ptr()),
      RE_INTER(Watrix33f*, intri_.data_ptr()),
      RE_INTER(Watrix34f*, w2c_.data_ptr()),
      RE_INTER(Wec2f*, bound_.data_ptr())
  );
}